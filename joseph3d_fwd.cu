#include "hip/hip_runtime.h"
#include "parallelproj.h"
#include "joseph3d_fwd_worker.h"
#include "debug.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void joseph3d_fwd_kernel(const float *xstart,
                                    const float *xend,
                                    const float *img,
                                    const float *img_origin,
                                    const float *voxsize,
                                    float *p,
                                    size_t nlors,
                                    const int *img_dim)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nlors)
    {
        joseph3d_fwd_worker(i, xstart, xend, img, img_origin, voxsize, p, img_dim);
    }
}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

void joseph3d_fwd(const float *xstart,
                  const float *xend,
                  const float *img,
                  const float *img_origin,
                  const float *voxsize,
                  float *p,
                  size_t nlors,
                  const int *img_dim,
                  int device_id,
                  int threadsperblock)
{

    const float *d_xstart = nullptr;
    const float *d_xend = nullptr;
    const float *d_img = nullptr;
    const float *d_img_origin = nullptr;
    const float *d_voxsize = nullptr;
    float *d_p = nullptr;
    const int *d_img_dim = nullptr;

    // get pointer attributes of all input and output arrays
    hipPointerAttribute_t xstart_attr;
    hipError_t err = hipPointerGetAttributes(&xstart_attr, xstart);
    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////
    // TODO get attributes of all other arrays
    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////

    bool needs_copy_back = false;
    bool is_cuda_managed_ptr = false;

    if (err == hipSuccess && (xstart_attr.type == hipMemoryTypeManaged))
    {
        is_cuda_managed_ptr = true;
        DEBUG_PRINT("Managed array is on device : %d\n", xstart_attr.device);
    }
    // else throw error
    else
    {
        needs_copy_back = true;
        throw std::runtime_error("Unsupported pointer type");
    }

    if (is_cuda_managed_ptr)
    {
        // all arrays are cuda malloc managed, so no need to copy to the device
        d_xstart = xstart;
        d_xend = xend;
        d_img = img;
        d_img_origin = img_origin;
        d_voxsize = voxsize;
        d_p = p;
        d_img_dim = img_dim;
    }
    else
    {
        DEBUG_PRINT("COPYING HOST TO DEVICE");
    }

    // get and print the current cuda device ID
    int current_device_id;
    hipGetDevice(&current_device_id);
    DEBUG_PRINT("Using CUDA device: %d\n", current_device_id);

    int num_blocks = (int)((nlors + threadsperblock - 1) / threadsperblock);
    joseph3d_fwd_kernel<<<num_blocks, threadsperblock>>>(d_xstart, d_xend, d_img,
                                                         d_img_origin, d_voxsize,
                                                         d_p, nlors, d_img_dim);
    hipDeviceSynchronize();

    // if (needs_copy_back) {
    //     hipMemcpy(array, device_array, size * sizeof(float), hipMemcpyDeviceToHost);
    //     hipFree(device_array);
    // }
}
