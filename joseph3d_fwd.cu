#include "hip/hip_runtime.h"
#include "parallelproj.h"
#include "joseph3d_fwd_worker.h"
#include "debug.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void joseph3d_fwd_kernel(const float *xstart,
                                    const float *xend,
                                    const float *img,
                                    const float *img_origin,
                                    const float *voxsize,
                                    float *p,
                                    size_t nlors,
                                    const int *img_dim)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nlors)
    {
        joseph3d_fwd_worker(i, xstart, xend, img, img_origin, voxsize, p, img_dim);
    }
}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

void joseph3d_fwd(const float *xstart,
                  const float *xend,
                  const float *img,
                  const float *img_origin,
                  const float *voxsize,
                  float *p,
                  size_t nlors,
                  const int *img_dim,
                  int device_id,
                  int threadsperblock)
{
    // Set the CUDA device
    if (device_id >= 0)
    {
        hipSetDevice(device_id);
    }

    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////
    // copy arrays to device if needed
    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////

    // handle xstart (read only)
    float *d_xstart = nullptr;
    hipPointerAttribute_t xstart_attr;
    hipError_t err_xstart = hipPointerGetAttributes(&xstart_attr, xstart);
    bool free_xstart = false;
    if (err_xstart == hipSuccess && (xstart_attr.type == hipMemoryTypeManaged))
    {
        hipMemPrefetchAsync(xstart, sizeof(float) * nlors * 3, device_id);
        hipMemAdvise(xstart, sizeof(float) * nlors * 3, hipMemAdviseSetReadMostly, device_id);
    }

    if (err_xstart == hipSuccess && (xstart_attr.type == hipMemoryTypeManaged ||
                                      xstart_attr.type == hipMemoryTypeDevice))
    {
        d_xstart = const_cast<float *>(xstart);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_xstart, sizeof(float) * nlors * 3);
        hipMemcpy(d_xstart, xstart, sizeof(float) * nlors * 3, hipMemcpyHostToDevice);
        free_xstart = true;
    }

    // handle xend (read only)
    float *d_xend = nullptr;
    hipPointerAttribute_t xend_attr;
    hipError_t err_xend = hipPointerGetAttributes(&xend_attr, xend);
    bool free_xend = false;
    if (err_xend == hipSuccess && xend_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(xend, sizeof(float) * nlors * 3, device_id);
        hipMemAdvise(xend, sizeof(float) * nlors * 3, hipMemAdviseSetReadMostly, device_id);
    }

    if (err_xend == hipSuccess && (xend_attr.type == hipMemoryTypeManaged ||
                                    xend_attr.type == hipMemoryTypeDevice))
    {
        d_xend = const_cast<float *>(xend);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_xend, sizeof(float) * nlors * 3);
        hipMemcpy(d_xend, xend, sizeof(float) * nlors * 3, hipMemcpyHostToDevice);
        free_xend = true;
    }

    // handle img (read only)
    float *d_img = nullptr;
    hipPointerAttribute_t img_attr;
    hipError_t err_img = hipPointerGetAttributes(&img_attr, img);
    bool free_img = false;
    if (err_img == hipSuccess && img_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(img, sizeof(float) * img_dim[0] * img_dim[1] * img_dim[2], device_id);
        hipMemAdvise(img, sizeof(float) * img_dim[0] * img_dim[1] * img_dim[2], hipMemAdviseSetReadMostly, device_id);
    }

    if (err_img == hipSuccess && (img_attr.type == hipMemoryTypeManaged ||
                                   img_attr.type == hipMemoryTypeDevice))
    {
        d_img = const_cast<float *>(img);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_img, sizeof(float) * img_dim[0] * img_dim[1] * img_dim[2]);
        hipMemcpy(d_img, img, sizeof(float) * img_dim[0] * img_dim[1] * img_dim[2], hipMemcpyHostToDevice);
        free_img = true;
    }

    // handle img_origin (read only)
    float *d_img_origin = nullptr;
    hipPointerAttribute_t img_origin_attr;
    hipError_t err_img_origin = hipPointerGetAttributes(&img_origin_attr, img_origin);
    bool free_img_origin = false;
    if (err_img_origin == hipSuccess && img_origin_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(img_origin, sizeof(float) * 3, device_id);
        hipMemAdvise(img_origin, sizeof(float) * 3, hipMemAdviseSetReadMostly, device_id);
    }

    if (err_img_origin == hipSuccess && (img_origin_attr.type == hipMemoryTypeManaged ||
                                          img_origin_attr.type == hipMemoryTypeDevice))
    {
        d_img_origin = const_cast<float *>(img_origin);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_img_origin, sizeof(float) * 3);
        hipMemcpy(d_img_origin, img_origin, sizeof(float) * 3, hipMemcpyHostToDevice);
        free_img_origin = true;
    }

    // handle voxsize (read only)
    float *d_voxsize = nullptr;
    hipPointerAttribute_t voxsize_attr;
    hipError_t err_voxsize = hipPointerGetAttributes(&voxsize_attr, voxsize);
    bool free_voxsize = false;
    if (err_voxsize == hipSuccess && voxsize_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(voxsize, sizeof(float) * 3, device_id);
        hipMemAdvise(voxsize, sizeof(float) * 3, hipMemAdviseSetReadMostly, device_id);
    }

    if (err_voxsize == hipSuccess && (voxsize_attr.type == hipMemoryTypeManaged ||
                                       voxsize_attr.type == hipMemoryTypeDevice))
    {
        d_voxsize = const_cast<float *>(voxsize);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_voxsize, sizeof(float) * 3);
        hipMemcpy(d_voxsize, voxsize, sizeof(float) * 3, hipMemcpyHostToDevice);
        free_voxsize = true;
    }

    // handle p (write)
    float *d_p = nullptr;
    hipPointerAttribute_t p_attr;
    hipError_t err_p = hipPointerGetAttributes(&p_attr, p);
    bool free_p = false;
    if (err_p == hipSuccess && p_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(p, sizeof(float) * nlors, device_id);
        hipMemAdvise(p, sizeof(float) * nlors, hipMemAdviseSetAccessedBy, device_id);
    }

    if (err_p == hipSuccess && (p_attr.type == hipMemoryTypeManaged ||
                                 p_attr.type == hipMemoryTypeDevice))
    {
        d_p = const_cast<float *>(p);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_p, sizeof(float) * nlors);
        hipMemcpy(d_p, p, sizeof(float) * nlors, hipMemcpyHostToDevice);
        free_p = true;
    }

    // handle img_dim (read only)
    int *d_img_dim = nullptr;
    hipPointerAttribute_t img_dim_attr;
    hipError_t err_img_dim = hipPointerGetAttributes(&img_dim_attr, img_dim);
    bool free_img_dim = false;
    if (err_img_dim == hipSuccess && img_dim_attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(img_dim, sizeof(int) * 3, device_id);
        hipMemAdvise(img_dim, sizeof(int) * 3, hipMemAdviseSetReadMostly, device_id);
    }

    if (err_img_dim == hipSuccess && (img_dim_attr.type == hipMemoryTypeManaged ||
                                       img_dim_attr.type == hipMemoryTypeDevice))
    {
        d_img_dim = const_cast<int *>(img_dim);
    }
    else
    {
        // host pointer case, transfer to device
        hipMalloc(&d_img_dim, sizeof(int) * 3);
        hipMemcpy(d_img_dim, img_dim, sizeof(int) * 3, hipMemcpyHostToDevice);
        free_img_dim = true;
    }

    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////
    // launch the kernel
    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////

#ifdef DEBUG
    // get and print the current cuda device ID
    int current_device_id;
    hipGetDevice(&current_device_id);
    DEBUG_PRINT("Using CUDA device: %d\n", current_device_id);
#endif

    int num_blocks = (int)((nlors + threadsperblock - 1) / threadsperblock);
    joseph3d_fwd_kernel<<<num_blocks, threadsperblock>>>(d_xstart, d_xend, d_img,
                                                         d_img_origin, d_voxsize,
                                                         d_p, nlors, d_img_dim);
    hipDeviceSynchronize();

    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////
    // free device memory if needed
    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////

    // Free device memory if it was allocated
    if (free_xstart)
        hipFree(d_xstart);
    if (free_xend)
        hipFree(d_xend);
    if (free_img)
        hipFree(d_img);
    if (free_img_origin)
        hipFree(d_img_origin);
    if (free_voxsize)
        hipFree(d_voxsize);
    if (free_p)
        hipFree(d_p);
    if (free_img_dim)
        hipFree(d_img_dim);
}
