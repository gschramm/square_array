#include "cuda_utils.h"
#include <iostream>

// Overload for constant input_ptr (const T*)
template <typename T>
void handle_cuda_input_array(const T *input_ptr, T **device_ptr, size_t size, bool &free_flag, int device_id, hipMemoryAdvise memory_hint)
{
    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, input_ptr);
    free_flag = false;

    if (err == hipSuccess && attr.type == hipMemoryTypeManaged)
    {
        // Prefetch and advise for managed memory
        hipMemPrefetchAsync(const_cast<void *>(static_cast<const void *>(input_ptr)), size, device_id);
        hipMemAdvise(const_cast<void *>(static_cast<const void *>(input_ptr)), size, memory_hint, device_id);
    }

    if (err == hipSuccess && (attr.type == hipMemoryTypeManaged || attr.type == hipMemoryTypeDevice))
    {
        // Assign managed or device pointer
        *device_ptr = const_cast<T *>(input_ptr);
    }
    else
    {
        // Host pointer case, transfer to device
        hipMalloc(device_ptr, size);
        hipMemcpy(*device_ptr, input_ptr, size, hipMemcpyHostToDevice);
        free_flag = true;
    }
}

// Overload for non-constant input_ptr (T*) (not const)
template <typename T>
void handle_cuda_input_array(T *input_ptr, T **device_ptr, size_t size, bool &free_flag, int device_id, hipMemoryAdvise memory_hint)
{
    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, input_ptr);
    free_flag = false;

    if (err == hipSuccess && attr.type == hipMemoryTypeManaged)
    {
        // Prefetch and advise for managed memory
        hipMemPrefetchAsync(input_ptr, size, device_id);
        hipMemAdvise(input_ptr, size, memory_hint, device_id);
    }

    if (err == hipSuccess && (attr.type == hipMemoryTypeManaged || attr.type == hipMemoryTypeDevice))
    {
        // Assign managed or device pointer
        *device_ptr = input_ptr;
    }
    else
    {
        // Host pointer case, transfer to device
        hipMalloc(device_ptr, size);
        hipMemcpy(*device_ptr, input_ptr, size, hipMemcpyHostToDevice);
        free_flag = true;
    }
}

// Explicit template instantiations
template void handle_cuda_input_array<double>(const double *, double **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<double>(double *, double **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<float>(const float *, float **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<float>(float *, float **, size_t, bool &, int, hipMemoryAdvise);

template void handle_cuda_input_array<int>(const int *, int **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<int>(int *, int **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<unsigned int>(const unsigned int *, unsigned int **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<unsigned int>(unsigned int *, unsigned int **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<size_t>(const size_t *, size_t **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<size_t>(size_t *, size_t **, size_t, bool &, int, hipMemoryAdvise);

template void handle_cuda_input_array<char>(const char *, char **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<char>(char *, char **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<unsigned char>(const unsigned char *, unsigned char **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<unsigned char>(unsigned char *, unsigned char **, size_t, bool &, int, hipMemoryAdvise);

template void handle_cuda_input_array<bool>(const bool *, bool **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<bool>(bool *, bool **, size_t, bool &, int, hipMemoryAdvise);
