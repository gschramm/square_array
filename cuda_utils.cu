#include "cuda_utils.h"

template <typename T>
void handle_cuda_input_array(const T *host_ptr, T **device_ptr, size_t size, bool &free_flag, int device_id, hipMemoryAdvise memory_hint)
{
    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, host_ptr);
    free_flag = false;

    if (err == hipSuccess && attr.type == hipMemoryTypeManaged)
    {
        hipMemPrefetchAsync(const_cast<void *>(static_cast<const void *>(host_ptr)), size, device_id);
        hipMemAdvise(const_cast<void *>(static_cast<const void *>(host_ptr)), size, memory_hint, device_id);
    }

    if (err == hipSuccess && (attr.type == hipMemoryTypeManaged || attr.type == hipMemoryTypeDevice))
    {
        *device_ptr = const_cast<T *>(host_ptr);
    }
    else
    {
        // Host pointer case, transfer to device
        hipMalloc(device_ptr, size);
        hipMemcpy(*device_ptr, host_ptr, size, hipMemcpyHostToDevice);
        free_flag = true;
    }
}

// Explicit template instantiations
template void handle_cuda_input_array<float>(const float *, float **, size_t, bool &, int, hipMemoryAdvise);
template void handle_cuda_input_array<int>(const int *, int **, size_t, bool &, int, hipMemoryAdvise);
