#include "hip/hip_runtime.h"
#include "square_array.h"
#include "square_op.h"
#include <hip/hip_runtime.h>

__global__ void square_kernel(float* data, size_t n, float* sum) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        compute_and_accumulate(data, idx, sum);
    }
}

void square_array(float* array, size_t size, float* result_sum) {
    float* device_array = nullptr;
    float* device_sum = nullptr;
    bool needs_copy_back = false;
    float zero = 0.0f;

    hipMalloc(&device_sum, sizeof(float));
    hipMemcpy(device_sum, &zero, sizeof(float), hipMemcpyHostToDevice);

    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, array);
    bool is_device_ptr = false;

#if CUDART_VERSION >= 10000
    if (err == hipSuccess && (attr.type == hipMemoryTypeDevice || attr.type == hipMemoryTypeManaged))
        is_device_ptr = true;
#else
    if (err == hipSuccess && attr.memoryType == hipMemoryTypeDevice)
        is_device_ptr = true;
#endif

    if (is_device_ptr) {
        device_array = array;
    } else {
        hipMalloc(&device_array, size * sizeof(float));
        hipMemcpy(device_array, array, size * sizeof(float), hipMemcpyHostToDevice);
        needs_copy_back = true;
    }

    int threads = 256;
    int blocks = (int)((size + threads - 1) / threads);
    square_kernel<<<blocks, threads>>>(device_array, size, device_sum);
    hipDeviceSynchronize();

    if (needs_copy_back) {
        hipMemcpy(array, device_array, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(device_array);
    }

    hipMemcpy(result_sum, device_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_sum);
}

