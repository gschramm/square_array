#include "hip/hip_runtime.h"
#include "square_array.h"
#include "square_op.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void square_kernel(float* data, size_t n, float* sum) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        compute_and_accumulate(data, idx, sum);
    }
}

void square_array(float* array, size_t size, float* result_sum, int device_id) {

    // the default device_id defined in the header is -1 (current active device)
    // if specified by the user, set the device_id (must be >= 0)
    if (device_id >= 0) {
        int device_count = 0;
        hipGetDeviceCount(&device_count);
    
        if (device_id >= device_count) {
            throw std::runtime_error("Invalid CUDA device ID: " + std::to_string(device_id));
        }
    
        hipSetDevice(device_id);
    }

    int current_device;
    hipGetDevice(&current_device);
    std::cout << "Using CUDA device " << current_device << std::endl;

    float* device_array = nullptr;
    float* device_sum = nullptr;
    bool needs_copy_back = false;
    float zero = 0.0f;

    hipMalloc(&device_sum, sizeof(float));
    hipMemcpy(device_sum, &zero, sizeof(float), hipMemcpyHostToDevice);

    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, array);
    bool is_device_ptr = false;

#if CUDART_VERSION >= 10000
    if (err == hipSuccess && (attr.type == hipMemoryTypeDevice || attr.type == hipMemoryTypeManaged))
        is_device_ptr = true;
#else
    if (err == hipSuccess && attr.memoryType == hipMemoryTypeDevice)
        is_device_ptr = true;
#endif

    if (is_device_ptr) {
        device_array = array;
    } else {
        hipMalloc(&device_array, size * sizeof(float));
        hipMemcpy(device_array, array, size * sizeof(float), hipMemcpyHostToDevice);
        needs_copy_back = true;
    }

    int threads = 256;
    int blocks = (int)((size + threads - 1) / threads);
    square_kernel<<<blocks, threads>>>(device_array, size, device_sum);
    hipDeviceSynchronize();

    if (needs_copy_back) {
        hipMemcpy(array, device_array, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(device_array);
    }

    hipMemcpy(result_sum, device_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_sum);
}

