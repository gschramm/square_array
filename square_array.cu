#include "hip/hip_runtime.h"
#include "square_array.h"
#include "square_op.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void square_kernel(float* data, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] = square_compute(data[idx]);
    }
}

void square_array(float* array, size_t size) {
    float* device_array = nullptr;
    bool needs_copy_back = false;

    // Detect pointer type
    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, array);
    bool is_device_ptr = false;

    if (err == hipSuccess) {
#if CUDART_VERSION >= 10000
        is_device_ptr = (attr.type == hipMemoryTypeDevice || attr.type == hipMemoryTypeManaged);
#else
        is_device_ptr = (attr.memoryType == hipMemoryTypeDevice);
#endif
    }

    if (is_device_ptr) {
        printf("input array is already on device\n");
        device_array = array;
    } else {
        // Allocate device memory and copy data
        printf("copying input array to device\n");
        hipMalloc(&device_array, size * sizeof(float));
        hipMemcpy(device_array, array, size * sizeof(float), hipMemcpyHostToDevice);
        needs_copy_back = true;
    }

    // Launch kernel
    int threads = 256;
    int blocks = static_cast<int>((size + threads - 1) / threads);
    square_kernel<<<blocks, threads>>>(device_array, size);
    hipDeviceSynchronize();

    // Copy result back if we allocated temp device memory
    if (needs_copy_back) {
        printf("copying result back to host\n");
        hipMemcpy(array, device_array, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(device_array);
    }
}

