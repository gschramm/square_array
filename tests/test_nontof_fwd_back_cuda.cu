#include "hip/hip_runtime.h"
#include "parallelproj.h"
#include "file_utils.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>

void test_cuda_managed_arrays();
void test_cuda_device_arrays();

int main()
{
    std::cout << "\n--- Testing with CUDA-Managed Arrays ---\n";
    test_cuda_managed_arrays();

    std::cout << "\n--- Testing with CUDA Device Arrays ---\n";
    test_cuda_device_arrays();

    return 0;
}

void test_cuda_managed_arrays()
{
    // CUDA-managed array test
    int h_img_dim[3] = {2, 3, 4};
    float h_voxsize[3] = {4.0f, 3.0f, 2.0f};

    float *cm_img_origin;
    hipMallocManaged(&cm_img_origin, 3 * sizeof(float));
    for (int i = 0; i < 3; ++i)
    {
        cm_img_origin[i] = (-(float)h_img_dim[i] / 2 + 0.5f) * h_voxsize[i];
    }

    std::vector<float> h_img = readArrayFromFile<float>("img.txt");
    float *cm_img;
    hipMallocManaged(&cm_img, h_img.size() * sizeof(float));
    std::copy(h_img.begin(), h_img.end(), cm_img);

    std::vector<float> h_vstart = readArrayFromFile<float>("vstart.txt");
    float *cm_vstart;
    hipMallocManaged(&cm_vstart, h_vstart.size() * sizeof(float));
    std::copy(h_vstart.begin(), h_vstart.end(), cm_vstart);

    std::vector<float> h_vend = readArrayFromFile<float>("vend.txt");
    float *cm_vend;
    hipMallocManaged(&cm_vend, h_vend.size() * sizeof(float));
    std::copy(h_vend.begin(), h_vend.end(), cm_vend);

    size_t nlors = h_vstart.size() / 3;

    float *cm_xstart, *cm_xend;
    hipMallocManaged(&cm_xstart, 3 * nlors * sizeof(float));
    hipMallocManaged(&cm_xend, 3 * nlors * sizeof(float));

    for (int ir = 0; ir < nlors; ir++)
    {
        for (int j = 0; j < 3; j++)
        {
            cm_xstart[ir * 3 + j] = cm_img_origin[j] + cm_vstart[ir * 3 + j] * h_voxsize[j];
            cm_xend[ir * 3 + j] = cm_img_origin[j] + cm_vend[ir * 3 + j] * h_voxsize[j];
        }
    }

    float *cm_img_fwd;
    hipMallocManaged(&cm_img_fwd, nlors * sizeof(float));
    joseph3d_fwd(cm_xstart, cm_xend, cm_img, cm_img_origin, h_voxsize, cm_img_fwd, nlors, h_img_dim, 0, 64);

    std::vector<float> h_expected_fwd_vals = readArrayFromFile<float>("expected_fwd_vals.txt");
    float fwd_diff = 0;
    float eps = 1e-7;

    for (int ir = 0; ir < nlors; ir++)
    {
        fwd_diff = std::abs(cm_img_fwd[ir] - h_expected_fwd_vals[ir]);
        if (fwd_diff > eps)
        {
            std::cerr << "CUDA-managed array test failed for ray " << ir << "\n";
            return;
        }
    }

    // Test the back projection
    float *cm_bimg;
    hipMallocManaged(&cm_bimg, h_img_dim[0] * h_img_dim[1] * h_img_dim[2] * sizeof(float));
    std::fill(cm_bimg, cm_bimg + (h_img_dim[0] * h_img_dim[1] * h_img_dim[2]), 0.0f);

    float *cm_ones;
    hipMallocManaged(&cm_ones, nlors * sizeof(float));
    std::fill(cm_ones, cm_ones + nlors, 1.0f);

    joseph3d_back(cm_xstart, cm_xend, cm_bimg, cm_img_origin, h_voxsize, cm_ones, nlors, h_img_dim);

    printf("\nCUDA-managed back projection of ones along all rays:\n");
    for (size_t i0 = 0; i0 < h_img_dim[0]; i0++)
    {
        for (size_t i1 = 0; i1 < h_img_dim[1]; i1++)
        {
            for (size_t i2 = 0; i2 < h_img_dim[2]; i2++)
            {
                printf("%.1f ", cm_bimg[h_img_dim[1] * h_img_dim[2] * i0 + h_img_dim[2] * i1 + i2]);
            }
            printf("\n");
        }
        printf("\n");
    }

    // Validate the back projection using adjointness
    float inner_product1 = 0.0f;
    float inner_product2 = 0.0f;

    for (size_t i = 0; i < h_img.size(); i++)
    {
        inner_product1 += cm_img[i] * cm_bimg[i];
    }

    for (size_t ir = 0; ir < nlors; ir++)
    {
        inner_product2 += cm_img_fwd[ir] * cm_ones[ir];
    }

    float ip_diff = fabs(inner_product1 - inner_product2);

    if (ip_diff > eps)
    {
        std::cerr << "CUDA-managed array back projection test failed: adjointness property violated.\n";
    }
    else
    {
        std::cout << "CUDA-managed array back projection test passed.\n";
    }

    hipFree(cm_img_origin);
    hipFree(cm_img);
    hipFree(cm_vstart);
    hipFree(cm_vend);
    hipFree(cm_xstart);
    hipFree(cm_xend);
    hipFree(cm_img_fwd);
    hipFree(cm_bimg);
    hipFree(cm_ones);
}

void test_cuda_device_arrays()
{
    // CUDA device array test
    int h_img_dim[3] = {2, 3, 4};
    float h_voxsize[3] = {4.0f, 3.0f, 2.0f};

    float h_img_origin[3];
    for (int i = 0; i < 3; ++i)
    {
        h_img_origin[i] = (-(float)h_img_dim[i] / 2 + 0.5f) * h_voxsize[i];
    }

    std::vector<float> h_img = readArrayFromFile<float>("img.txt");
    float *d_img;
    hipMalloc(&d_img, h_img.size() * sizeof(float));
    hipMemcpy(d_img, h_img.data(), h_img.size() * sizeof(float), hipMemcpyHostToDevice);

    std::vector<float> h_vstart = readArrayFromFile<float>("vstart.txt");
    float *d_vstart;
    hipMalloc(&d_vstart, h_vstart.size() * sizeof(float));
    hipMemcpy(d_vstart, h_vstart.data(), h_vstart.size() * sizeof(float), hipMemcpyHostToDevice);

    std::vector<float> h_vend = readArrayFromFile<float>("vend.txt");
    float *d_vend;
    hipMalloc(&d_vend, h_vend.size() * sizeof(float));
    hipMemcpy(d_vend, h_vend.data(), h_vend.size() * sizeof(float), hipMemcpyHostToDevice);

    size_t nlors = h_vstart.size() / 3;

    float *d_xstart, *d_xend;
    hipMalloc(&d_xstart, 3 * nlors * sizeof(float));
    hipMalloc(&d_xend, 3 * nlors * sizeof(float));

    for (int ir = 0; ir < nlors; ir++)
    {
        for (int j = 0; j < 3; j++)
        {
            float xstart_val = h_img_origin[j] + h_vstart[ir * 3 + j] * h_voxsize[j];
            float xend_val = h_img_origin[j] + h_vend[ir * 3 + j] * h_voxsize[j];
            hipMemcpy(&d_xstart[ir * 3 + j], &xstart_val, sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(&d_xend[ir * 3 + j], &xend_val, sizeof(float), hipMemcpyHostToDevice);
        }
    }

    float *d_img_fwd;
    hipMalloc(&d_img_fwd, nlors * sizeof(float));
    joseph3d_fwd(d_xstart, d_xend, d_img, h_img_origin, h_voxsize, d_img_fwd, nlors, h_img_dim, 0, 64);

    std::vector<float> h_img_fwd(nlors);
    hipMemcpy(h_img_fwd.data(), d_img_fwd, nlors * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> h_expected_fwd_vals = readArrayFromFile<float>("expected_fwd_vals.txt");
    float fwd_diff = 0;
    float eps = 1e-7;

    for (int ir = 0; ir < nlors; ir++)
    {
        fwd_diff = std::abs(h_img_fwd[ir] - h_expected_fwd_vals[ir]);
        if (fwd_diff > eps)
        {
            std::cerr << "CUDA device array test failed for ray " << ir << "\n";
            return;
        }
    }

    // Test the back projection
    float *d_bimg;
    hipMalloc(&d_bimg, h_img_dim[0] * h_img_dim[1] * h_img_dim[2] * sizeof(float));
    hipMemset(d_bimg, 0, h_img_dim[0] * h_img_dim[1] * h_img_dim[2] * sizeof(float));

    float *d_ones;
    hipMalloc(&d_ones, nlors * sizeof(float));
    std::vector<float> h_ones(nlors, 1.0f);
    hipMemcpy(d_ones, h_ones.data(), nlors * sizeof(float), hipMemcpyHostToDevice);

    joseph3d_back(d_xstart, d_xend, d_bimg, h_img_origin, h_voxsize, d_ones, nlors, h_img_dim);

    std::vector<float> h_bimg(h_img_dim[0] * h_img_dim[1] * h_img_dim[2]);
    hipMemcpy(h_bimg.data(), d_bimg, h_bimg.size() * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nCUDA device back projection of ones along all rays:\n");
    for (size_t i0 = 0; i0 < h_img_dim[0]; i0++)
    {
        for (size_t i1 = 0; i1 < h_img_dim[1]; i1++)
        {
            for (size_t i2 = 0; i2 < h_img_dim[2]; i2++)
            {
                printf("%.1f ", h_bimg[h_img_dim[1] * h_img_dim[2] * i0 + h_img_dim[2] * i1 + i2]);
            }
            printf("\n");
        }
        printf("\n");
    }

    // Validate the back projection using adjointness
    float inner_product1 = 0.0f;
    float inner_product2 = 0.0f;

    for (size_t i = 0; i < h_img.size(); i++)
    {
        inner_product1 += h_img[i] * h_bimg[i];
    }

    for (size_t ir = 0; ir < nlors; ir++)
    {
        inner_product2 += h_img_fwd[ir] * h_ones[ir];
    }

    float ip_diff = fabs(inner_product1 - inner_product2);

    if (ip_diff > eps)
    {
        std::cerr << "CUDA device array back projection test failed: adjointness property violated.\n";
    }
    else
    {
        std::cout << "CUDA device array back projection test passed.\n";
    }

    hipFree(d_img);
    hipFree(d_vstart);
    hipFree(d_vend);
    hipFree(d_xstart);
    hipFree(d_xend);
    hipFree(d_img_fwd);
    hipFree(d_bimg);
    hipFree(d_ones);
}
